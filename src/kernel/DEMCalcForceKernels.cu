#include "hip/hip_runtime.h"
// DEM force computation related custom kernels
#include <DEM/Defines.h>
#include <DEMHelperKernels.cuh>
#include <DEMCollisionKernels.cuh>
_kernelIncludes_;

// If clump templates are jitified, they will be below
_clumpTemplateDefs_;
// Definitions of analytical entites are below
_analyticalEntityDefs_;
// Material properties are below
_materialDefs_;
// If mass properties are jitified, then they are below
_massDefs_;
_moiDefs_;
// If the user has some utility functions, they will be included here
_forceModelPrerequisites_;

template <typename T1>
inline __device__ void equipOwnerPosRot(deme::DEMSimParams* simParams,
                                        deme::DEMDataDT* granData,
                                        const deme::bodyID_t& myOwner,
                                        T1& relPos,
                                        double3& ownerPos,
                                        double3& bodyPos,
                                        float4& oriQ) {
    voxelIDToPosition<double, deme::voxelID_t, deme::subVoxelPos_t>(
        ownerPos.x, ownerPos.y, ownerPos.z, granData->voxelID[myOwner], granData->locX[myOwner],
        granData->locY[myOwner], granData->locZ[myOwner], _nvXp2_, _nvYp2_, _voxelSize_, _l_);
    // Do this and we get the `true' pos...
    ownerPos.x += simParams->LBFX;
    ownerPos.y += simParams->LBFY;
    ownerPos.z += simParams->LBFZ;
    oriQ.w = granData->oriQw[myOwner];
    oriQ.x = granData->oriQx[myOwner];
    oriQ.y = granData->oriQy[myOwner];
    oriQ.z = granData->oriQz[myOwner];
    applyOriQToVector3(relPos.x, relPos.y, relPos.z, oriQ.w, oriQ.x, oriQ.y, oriQ.z);
    bodyPos.x = ownerPos.x + (double)relPos.x;
    bodyPos.y = ownerPos.y + (double)relPos.y;
    bodyPos.z = ownerPos.z + (double)relPos.z;
}

// Template device function for contact force calculation - will be called by 5 specialized kernels
template <deme::contact_t CONTACT_TYPE>
__device__ __forceinline__ void calculateContactForcesImpl(deme::DEMSimParams* simParams,
                                                           deme::DEMDataDT* granData,
                                                           deme::contactPairs_t myContactID) {
    // Contact type is known at compile time
    deme::contact_t ContactType = CONTACT_TYPE;
    // The following quantities are always calculated, regardless of force model
    double3 contactPnt;
    float3 B2A;  // Unit vector pointing from body B to body A (contact normal)
    double overlapDepth;
    double3 AOwnerPos, bodyAPos, BOwnerPos, bodyBPos;
    float AOwnerMass, ARadius, BOwnerMass, BRadius;
    float4 AOriQ, BOriQ;
    deme::materialsOffset_t bodyAMatType, bodyBMatType;
    // The user-specified extra margin size (how much we should be lenient in determining `in-contact')
    float extraMarginSize = 0.;
    // Triangle A's three points are defined outside, as may be reused in B's acquisition and penetration calc.
    double3 triANode1, triANode2, triANode3;
    // Then allocate the optional quantities that will be needed in the force model (note: this one can't be in a
    // curly bracket, obviously...)
    _forceModelIngredientDefinition_;
    // Take care of 2 bodies in order, bodyA first, grab location and velocity to local cache
    // Decompose ContactType to get the types of A and B (known at compile time)
    constexpr deme::geoType_t AType = (CONTACT_TYPE >> 4);
    constexpr deme::geoType_t BType = (CONTACT_TYPE & 0xF);

        // ----------------------------------------------------------------
        // Based on A's type, equip info
        // ----------------------------------------------------------------
        if (AType == deme::GEO_T_SPHERE) {
            deme::bodyID_t sphereID = granData->idGeometryA[myContactID];
            deme::bodyID_t myOwner = granData->ownerClumpBody[sphereID];

            float3 myRelPos;
            float myRadius;
            // Get my component offset info from either jitified arrays or global memory
            // Outputs myRelPos, myRadius
            // Use an input named exactly `sphereID' which is the id of this sphere component
            { _componentAcqStrat_; }

            // Get my mass info from either jitified arrays or global memory
            // Outputs myMass
            // Use an input named exactly `myOwner' which is the id of this owner
            {
                float myMass;
                _massAcqStrat_;
                AOwnerMass = myMass;
            }

            // Optional force model ingredients are loaded here...
            _forceModelIngredientAcqForA_;
            _forceModelGeoWildcardAcqForASph_;

            equipOwnerPosRot(simParams, granData, myOwner, myRelPos, AOwnerPos, bodyAPos, AOriQ);

            ARadius = myRadius;
            bodyAMatType = granData->sphereMaterialOffset[sphereID];
            extraMarginSize = granData->familyExtraMarginSize[AOwnerFamily];
        } else if (AType == deme::GEO_T_TRIANGLE) {
            // Geometry ID here is called sphereID, although it is not a sphere, it's more like triID. But naming it
            // sphereID makes the acquisition process cleaner.
            deme::bodyID_t sphereID = granData->idGeometryA[myContactID];
            deme::bodyID_t myOwner = granData->ownerMesh[sphereID];
            //// TODO: Is this OK?
            ARadius = DEME_HUGE_FLOAT;
            bodyAMatType = granData->triMaterialOffset[sphereID];

            // As the grace margin, the distance (negative overlap) just needs to be within the grace margin. So we pick
            // the larger of the 2 familyExtraMarginSize.
            extraMarginSize = granData->familyExtraMarginSize[AOwnerFamily];

            triANode1 = to_double3(granData->relPosNode1[sphereID]);
            triANode2 = to_double3(granData->relPosNode2[sphereID]);
            triANode3 = to_double3(granData->relPosNode3[sphereID]);

            // Get my mass info from either jitified arrays or global memory
            // Outputs myMass
            // Use an input named exactly `myOwner' which is the id of this owner
            {
                float myMass;
                _massAcqStrat_;
                AOwnerMass = myMass;
            }
            _forceModelIngredientAcqForA_;
            _forceModelGeoWildcardAcqForATri_;

            // bodyAPos is for a place holder for the outcome triANode1 position
            equipOwnerPosRot(simParams, granData, myOwner, triANode1, AOwnerPos, bodyAPos, AOriQ);
            triANode1 = bodyAPos;
            // Do this to node 2 and 3 as well
            applyOriQToVector3(triANode2.x, triANode2.y, triANode2.z, AOriQ.w, AOriQ.x, AOriQ.y, AOriQ.z);
            triANode2 += AOwnerPos;
            applyOriQToVector3(triANode3.x, triANode3.y, triANode3.z, AOriQ.w, AOriQ.x, AOriQ.y, AOriQ.z);
            triANode3 += AOwnerPos;
            // Assign the correct bodyAPos
            bodyAPos = triangleCentroid<double3>(triANode1, triANode2, triANode3);
        } else {
            // Currently, we only support sphere and mesh for body A
            ContactType = deme::NOT_A_CONTACT;
        }

        // ----------------------------------------------------------------
        // Then B, location and velocity, depending on type
        // ----------------------------------------------------------------
        if (BType == deme::GEO_T_SPHERE) {
            deme::bodyID_t sphereID = granData->idGeometryB[myContactID];
            deme::bodyID_t myOwner = granData->ownerClumpBody[sphereID];

            float3 myRelPos;
            float myRadius;
            // Get my component offset info from either jitified arrays or global memory
            // Outputs myRelPos, myRadius
            // Use an input named exactly `sphereID' which is the id of this sphere component
            { _componentAcqStrat_; }

            // Get my mass info from either jitified arrays or global memory
            // Outputs myMass
            // Use an input named exactly `myOwner' which is the id of this owner
            {
                float myMass;
                _massAcqStrat_;
                BOwnerMass = myMass;
            }
            _forceModelIngredientAcqForB_;
            _forceModelGeoWildcardAcqForBSph_;

            equipOwnerPosRot(simParams, granData, myOwner, myRelPos, BOwnerPos, bodyBPos, BOriQ);

            BRadius = myRadius;
            bodyBMatType = granData->sphereMaterialOffset[sphereID];

            // As the grace margin, the distance (negative overlap) just needs to be within the grace margin. So we pick
            // the larger of the 2 familyExtraMarginSize.
            extraMarginSize = (extraMarginSize > granData->familyExtraMarginSize[BOwnerFamily])
                                  ? extraMarginSize
                                  : granData->familyExtraMarginSize[BOwnerFamily];

            // If B is a sphere, then A can only be a sphere
            checkSpheresOverlap<double, float>(bodyAPos.x, bodyAPos.y, bodyAPos.z, ARadius, bodyBPos.x, bodyBPos.y,
                                               bodyBPos.z, BRadius, contactPnt.x, contactPnt.y, contactPnt.z, B2A.x,
                                               B2A.y, B2A.z, overlapDepth);
            // If overlapDepth is negative then it might still be considered in contact, if the extra margins of A and B
            // combined is larger than abs(overlapDepth)
            if (overlapDepth < -extraMarginSize) {
                ContactType = deme::NOT_A_CONTACT;
            }

        } else if (BType == deme::GEO_T_TRIANGLE) {
            // Geometry ID here is called sphereID, although it is not a sphere, it's more like triID. But naming it
            // sphereID makes the acquisition process cleaner.
            deme::bodyID_t sphereID = granData->idGeometryB[myContactID];
            deme::bodyID_t myOwner = granData->ownerMesh[sphereID];
            //// TODO: Is this OK?
            BRadius = DEME_HUGE_FLOAT;
            bodyBMatType = granData->triMaterialOffset[sphereID];

            // As the grace margin, the distance (negative overlap) just needs to be within the grace margin. So we pick
            // the larger of the 2 familyExtraMarginSize.
            extraMarginSize = (extraMarginSize > granData->familyExtraMarginSize[BOwnerFamily])
                                  ? extraMarginSize
                                  : granData->familyExtraMarginSize[BOwnerFamily];
            // extraMarginSize here is purely family-based extra margin, so it can be used to determine if the user
            // potentially needs remote (non-contact) force calculation.
            bool needsNonContactPenetrationCalc = (extraMarginSize > 0.);

            double3 triBNode1 = to_double3(granData->relPosNode1[sphereID]);
            double3 triBNode2 = to_double3(granData->relPosNode2[sphereID]);
            double3 triBNode3 = to_double3(granData->relPosNode3[sphereID]);

            // Get my mass info from either jitified arrays or global memory
            // Outputs myMass
            // Use an input named exactly `myOwner' which is the id of this owner
            {
                float myMass;
                _massAcqStrat_;
                BOwnerMass = myMass;
            }
            _forceModelIngredientAcqForB_;
            _forceModelGeoWildcardAcqForBTri_;

            // bodyBPos is for a place holder for the outcome triBNode1 position
            equipOwnerPosRot(simParams, granData, myOwner, triBNode1, BOwnerPos, bodyBPos, BOriQ);
            triBNode1 = bodyBPos;
            // Do this to node 2 and 3 as well
            applyOriQToVector3(triBNode2.x, triBNode2.y, triBNode2.z, BOriQ.w, BOriQ.x, BOriQ.y, BOriQ.z);
            triBNode2 += BOwnerPos;
            applyOriQToVector3(triBNode3.x, triBNode3.y, triBNode3.z, BOriQ.w, BOriQ.x, BOriQ.y, BOriQ.z);
            triBNode3 += BOwnerPos;
            // Assign the correct bodyBPos
            bodyBPos = triangleCentroid<double3>(triBNode1, triBNode2, triBNode3);

            // If B is a triangle, then A can be a sphere or a triangle. But this branching is not too bad, as most
            // threads in this block will have the same ContactType.
            if (AType == deme::GEO_T_SPHERE) {
                double3 contact_normal;
                bool in_contact = checkTriSphereOverlap<double3, double>(
                    triBNode1, triBNode2, triBNode3, bodyAPos, ARadius, contact_normal, overlapDepth, contactPnt);
                B2A = to_float3(contact_normal);

                // Sphere--triangle is a bit tricky. Extra margin should only take effect when it comes from the
                // positive direction of the mesh facet. If not, sphere-setting-on-needle case will give huge
                // penetration since in that case, overlapDepth is very negative and this will be considered in-contact.
                // So the cases we exclude are: too far away while at the positive direction; not in contact while at
                // the negative side.
                // Also checkTriSphereOverlap gives positive number for overlapping cases
                if ((overlapDepth < -extraMarginSize) || (!in_contact && overlapDepth > 0.)) {
                    ContactType = deme::NOT_A_CONTACT;
                }
            } else if (AType == deme::GEO_T_TRIANGLE) {
                // Triangle--triangle contact, a bit more complex...
                double3 contact_normal;
                checkTriangleTriangleOverlap<double3, double>(triANode1, triANode2, triANode3, triBNode1, triBNode2,
                                                              triBNode3, contact_normal, overlapDepth, contactPnt,
                                                              needsNonContactPenetrationCalc);
                B2A = to_float3(contact_normal);
                // Fix ContactType if needed
                if (overlapDepth < -extraMarginSize) {
                    ContactType = deme::NOT_A_CONTACT;
                }
            }

        } else if (BType == deme::GEO_T_ANALYTICAL) {
            // Geometry ID here is called sphereID, although it is not a sphere, it's more like analyticalID. But naming
            // it sphereID makes the acquisition process cleaner.
            deme::objID_t sphereID = granData->idGeometryB[myContactID];
            deme::bodyID_t myOwner = objOwner[sphereID];
            // If B is analytical entity, its owner, relative location, material info is jitified.
            bodyBMatType = objMaterial[sphereID];
            BOwnerMass = objMass[sphereID];
            //// TODO: Is this OK?
            BRadius = DEME_HUGE_FLOAT;
            float3 myRelPos;
            float3 bodyBRot;
            myRelPos.x = objRelPosX[sphereID];
            myRelPos.y = objRelPosY[sphereID];
            myRelPos.z = objRelPosZ[sphereID];
            _forceModelIngredientAcqForB_;
            _forceModelGeoWildcardAcqForBAnal_;

            equipOwnerPosRot(simParams, granData, myOwner, myRelPos, BOwnerPos, bodyBPos, BOriQ);

            // As the grace margin, the distance (negative overlap) just needs to be within the grace margin. So we pick
            // the larger of the 2 familyExtraMarginSize.
            extraMarginSize = (extraMarginSize > granData->familyExtraMarginSize[BOwnerFamily])
                                  ? extraMarginSize
                                  : granData->familyExtraMarginSize[BOwnerFamily];

            // B's orientation (such as plane normal) is rotated with its owner too
            bodyBRot.x = objRotX[sphereID];
            bodyBRot.y = objRotY[sphereID];
            bodyBRot.z = objRotZ[sphereID];
            applyOriQToVector3<float, deme::oriQ_t>(bodyBRot.x, bodyBRot.y, bodyBRot.z, BOriQ.w, BOriQ.x, BOriQ.y,
                                                    BOriQ.z);

            // If B is an analytical entity, then A can be a sphere or a triangle. But this branching is not too bad, as
            // most threads in this block will have the same ContactType.
            if (AType == deme::GEO_T_SPHERE) {
                // Note for this test on dT side we don't enlarge entities
                checkSphereEntityOverlap<double3, float, double>(
                    bodyAPos, ARadius, objType[sphereID], bodyBPos, bodyBRot, objSize1[sphereID], objSize2[sphereID],
                    objSize3[sphereID], objNormal[sphereID], 0.0, contactPnt, B2A, overlapDepth);
                // Fix ContactType if needed
                if (overlapDepth < -extraMarginSize) {
                    ContactType = deme::NOT_A_CONTACT;
                }
            } else if (AType == deme::GEO_T_TRIANGLE) {
                calcTriEntityOverlap<double3, double>(
                    triANode1, triANode2, triANode3, objType[sphereID], bodyBPos, bodyBRot, objSize1[sphereID],
                    objSize2[sphereID], objSize3[sphereID], objNormal[sphereID], contactPnt, B2A, overlapDepth);
                // Fix ContactType if needed
                if (overlapDepth < -extraMarginSize) {
                    ContactType = deme::NOT_A_CONTACT;
                }
            }
        }

        _forceModelContactWildcardAcq_;
        if (ContactType != deme::NOT_A_CONTACT) {
            float3 force = make_float3(0, 0, 0);
            float3 torque_only_force = make_float3(0, 0, 0);
            // Local position of the contact point is always a piece of info we require... regardless of force model
            float3 locCPA = to_float3(contactPnt - AOwnerPos);
            float3 locCPB = to_float3(contactPnt - BOwnerPos);
            // Now map this contact point location to bodies' local ref
            applyOriQToVector3<float, deme::oriQ_t>(locCPA.x, locCPA.y, locCPA.z, AOriQ.w, -AOriQ.x, -AOriQ.y,
                                                    -AOriQ.z);
            applyOriQToVector3<float, deme::oriQ_t>(locCPB.x, locCPB.y, locCPB.z, BOriQ.w, -BOriQ.x, -BOriQ.y,
                                                    -BOriQ.z);
            // The following part, the force model, is user-specifiable
            // NOTE!! "force" and all wildcards must be properly set by this piece of code
            { _DEMForceModel_; }

            // Write contact location values back to global memory
            _contactInfoWrite_;

            // If force model modifies owner wildcards, write them back here
            _forceModelOwnerWildcardWrite_;

            // Optionally, the forces can be reduced to acc right here (may be faster)
            _forceCollectInPlaceStrat_;
        } else {
            // The contact is no longer active, so we need to destroy its contact history recording
            _forceModelContactWildcardDestroy_;
        }

        // Updated contact wildcards need to be write back to global mem. It is here because contact wildcard may need
        // to be destroyed for non-contact, so it has to go last.
        _forceModelContactWildcardWrite_;
}

// 5 specialized kernels for different contact types
__global__ void calculateContactForces_SphSph(deme::DEMSimParams* simParams,
                                              deme::DEMDataDT* granData,
                                              size_t startOffset,
                                              size_t nContactPairs) {
    deme::contactPairs_t myContactID = startOffset + blockIdx.x * blockDim.x + threadIdx.x;
    if (myContactID < startOffset + nContactPairs) {
        calculateContactForcesImpl<deme::SPHERE_SPHERE_CONTACT>(simParams, granData, myContactID);
    }
}

__global__ void calculateContactForces_SphTri(deme::DEMSimParams* simParams,
                                              deme::DEMDataDT* granData,
                                              size_t startOffset,
                                              size_t nContactPairs) {
    deme::contactPairs_t myContactID = startOffset + blockIdx.x * blockDim.x + threadIdx.x;
    if (myContactID < startOffset + nContactPairs) {
        calculateContactForcesImpl<deme::SPHERE_TRIANGLE_CONTACT>(simParams, granData, myContactID);
    }
}

__global__ void calculateContactForces_SphAnal(deme::DEMSimParams* simParams,
                                               deme::DEMDataDT* granData,
                                               size_t startOffset,
                                               size_t nContactPairs) {
    deme::contactPairs_t myContactID = startOffset + blockIdx.x * blockDim.x + threadIdx.x;
    if (myContactID < startOffset + nContactPairs) {
        calculateContactForcesImpl<deme::SPHERE_ANALYTICAL_CONTACT>(simParams, granData, myContactID);
    }
}

__global__ void calculateContactForces_TriTri(deme::DEMSimParams* simParams,
                                              deme::DEMDataDT* granData,
                                              size_t startOffset,
                                              size_t nContactPairs) {
    deme::contactPairs_t myContactID = startOffset + blockIdx.x * blockDim.x + threadIdx.x;
    if (myContactID < startOffset + nContactPairs) {
        calculateContactForcesImpl<deme::TRIANGLE_TRIANGLE_CONTACT>(simParams, granData, myContactID);
    }
}

__global__ void calculateContactForces_TriAnal(deme::DEMSimParams* simParams,
                                               deme::DEMDataDT* granData,
                                               size_t startOffset,
                                               size_t nContactPairs) {
    deme::contactPairs_t myContactID = startOffset + blockIdx.x * blockDim.x + threadIdx.x;
    if (myContactID < startOffset + nContactPairs) {
        calculateContactForcesImpl<deme::TRIANGLE_ANALYTICAL_CONTACT>(simParams, granData, myContactID);
    }
}
