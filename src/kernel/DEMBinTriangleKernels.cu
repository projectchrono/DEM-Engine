#include "hip/hip_runtime.h"
// DEM bin--sphere relations-related custom kernels
#include <DEM/Defines.h>
#include <DEMCollisionKernels_SphTri_TriTri.cuh>
#include <DEMTriangleBoxIntersect.cu>
_kernelIncludes_;

// Definitions of analytical entites are below
_analyticalEntityDefs_;

inline __device__ float3
sandwichVertex(float3 vertex, const float3& incenter, const float3& side, const float3& normal, float beta) {
    // The vector along which we enlarge the triangle
    float3 expandVec = normalize(vertex - incenter);

    // Use a side starting from the vertex and the vector from the vertex to the incenter to figure out the half angle
    const float cos_halfangle = dot(-expandVec, side) / length(side);
    // Then the distance to advance the vertex along the expand vector...
    const float enlarge_dist = beta / sqrt(1. - cos_halfangle * cos_halfangle);

    vertex += expandVec * enlarge_dist;
    vertex += normal * beta;
    return vertex;
}

__global__ void makeTriangleSandwich(deme::DEMSimParams* simParams,
                                     deme::DEMDataKT* granData,
                                     float3* sandwichANode1,
                                     float3* sandwichANode2,
                                     float3* sandwichANode3,
                                     float3* sandwichBNode1,
                                     float3* sandwichBNode2,
                                     float3* sandwichBNode3) {
    deme::bodyID_t triID = blockIdx.x * blockDim.x + threadIdx.x;
    if (triID < simParams->nTriGM) {
        // Get my component offset info from global array
        const float3 p1 = granData->relPosNode1[triID];
        const float3 p2 = granData->relPosNode2[triID];
        const float3 p3 = granData->relPosNode3[triID];
        const deme::bodyID_t myOwnerID = granData->ownerMesh[triID];

        // Get the incenter of this triangle.
        // This is because we use the incenter to enalrge a triangle. See for example, this
        // https://stackoverflow.com/questions/36554898/algorithm-for-putting-double-border-around-isosceles-triangle.
        const float3 incenter = triangleIncenter<float3>(p1, p2, p3);
        // Generate normal using RHR from nodes 1, 2, and 3
        float3 triNormal = face_normal<float3>(p1, p2, p3);

        sandwichANode1[triID] = sandwichVertex(p1, incenter, p2 - p1, triNormal, granData->marginSize[myOwnerID]);
        sandwichANode2[triID] = sandwichVertex(p2, incenter, p3 - p2, triNormal, granData->marginSize[myOwnerID]);
        sandwichANode3[triID] = sandwichVertex(p3, incenter, p1 - p3, triNormal, granData->marginSize[myOwnerID]);
        // The other sandwich triangle needs to have an opposite normal direction
        sandwichBNode1[triID] = sandwichVertex(p1, incenter, p2 - p1, -triNormal, granData->marginSize[myOwnerID]);
        sandwichBNode2[triID] = sandwichVertex(p3, incenter, p1 - p3, -triNormal, granData->marginSize[myOwnerID]);
        sandwichBNode3[triID] = sandwichVertex(p2, incenter, p3 - p2, -triNormal, granData->marginSize[myOwnerID]);
    }
}

inline __device__ void figureOutNodeAndBoundingBox(deme::DEMSimParams* simParams,
                                                   deme::DEMDataKT* granData,
                                                   const deme::bodyID_t& triID,
                                                   float3& vA,
                                                   float3& vB,
                                                   float3& vC,
                                                   deme::binID_t L[],
                                                   deme::binID_t U[],
                                                   float3 loc_vA,
                                                   float3 loc_vB,
                                                   float3 loc_vC) {
    // My sphere voxel ID and my relPos
    deme::bodyID_t myOwnerID = granData->ownerMesh[triID];

    double3 ownerXYZ;
    voxelIDToPosition<double, deme::voxelID_t, deme::subVoxelPos_t>(
        ownerXYZ.x, ownerXYZ.y, ownerXYZ.z, granData->voxelID[myOwnerID], granData->locX[myOwnerID],
        granData->locY[myOwnerID], granData->locZ[myOwnerID], _nvXp2_, _nvYp2_, _voxelSize_, _l_);
    const float myOriQw = granData->oriQw[myOwnerID];
    const float myOriQx = granData->oriQx[myOwnerID];
    const float myOriQy = granData->oriQy[myOwnerID];
    const float myOriQz = granData->oriQz[myOwnerID];
    applyOriQToVector3<float, deme::oriQ_t>(loc_vA.x, loc_vA.y, loc_vA.z, myOriQw, myOriQx, myOriQy, myOriQz);
    applyOriQToVector3<float, deme::oriQ_t>(loc_vB.x, loc_vB.y, loc_vB.z, myOriQw, myOriQx, myOriQy, myOriQz);
    applyOriQToVector3<float, deme::oriQ_t>(loc_vC.x, loc_vC.y, loc_vC.z, myOriQw, myOriQx, myOriQy, myOriQz);
    vA = ownerXYZ + loc_vA;
    vB = ownerXYZ + loc_vB;
    vC = ownerXYZ + loc_vC;

    boundingBoxIntersectBin(L, U, vA, vB, vC, simParams);
}

__global__ void getNumberOfBinsEachTriangleTouches(deme::DEMSimParams* simParams,
                                                   deme::DEMDataKT* granData,
                                                   deme::binsTriangleTouches_t* numBinsTriTouches,
                                                   deme::objID_t* numAnalGeoTriTouches,
                                                   float3* nodeA1,
                                                   float3* nodeB1,
                                                   float3* nodeC1,
                                                   float3* nodeA2,
                                                   float3* nodeB2,
                                                   float3* nodeC2,
                                                   bool meshUniversalContact) {
    deme::bodyID_t triID = blockIdx.x * blockDim.x + threadIdx.x;

    if (triID < simParams->nTriGM) {
        // 3 vertices of the triangle, in true space location but without adding LBF point (since purely voxel- and
        // bin-based locations don't need that)
        float3 vA1, vB1, vC1, vA2, vB2, vC2;
        deme::binID_t L1[3], L2[3], U1[3], U2[3];
        figureOutNodeAndBoundingBox(simParams, granData, triID, vA1, vB1, vC1, L1, U1, nodeA1[triID], nodeB1[triID],
                                    nodeC1[triID]);
        figureOutNodeAndBoundingBox(simParams, granData, triID, vA2, vB2, vC2, L2, U2, nodeA2[triID], nodeB2[triID],
                                    nodeC2[triID]);
        L1[0] = DEME_MIN(L1[0], L2[0]);
        L1[1] = DEME_MIN(L1[1], L2[1]);
        L1[2] = DEME_MIN(L1[2], L2[2]);
        U1[0] = DEME_MAX(U1[0], U2[0]);
        U1[1] = DEME_MAX(U1[1], U2[1]);
        U1[2] = DEME_MAX(U1[2], U2[2]);

        unsigned int numSDsTouched = 0;
        // Triangle may span a collection of bins...
        // BTW, I don't know why Chrono::GPU had to check the so-called 3 cases, and create thread divergence like that.
        // Just sweep through all potential bins and you are fine.
        float BinCenter[3];
        float BinHalfSizes[3];
        BinHalfSizes[0] = simParams->binSize / 2. + DEME_BIN_ENLARGE_RATIO_FOR_FACETS * simParams->binSize;
        BinHalfSizes[1] = simParams->binSize / 2. + DEME_BIN_ENLARGE_RATIO_FOR_FACETS * simParams->binSize;
        BinHalfSizes[2] = simParams->binSize / 2. + DEME_BIN_ENLARGE_RATIO_FOR_FACETS * simParams->binSize;
        for (deme::binID_t i = L1[0]; i <= U1[0]; i++) {
            for (deme::binID_t j = L1[1]; j <= U1[1]; j++) {
                for (deme::binID_t k = L1[2]; k <= U1[2]; k++) {
                    BinCenter[0] = simParams->binSize * i + simParams->binSize / 2.;
                    BinCenter[1] = simParams->binSize * j + simParams->binSize / 2.;
                    BinCenter[2] = simParams->binSize * k + simParams->binSize / 2.;

                    if (check_TriangleBoxOverlap(BinCenter, BinHalfSizes, vA1, vB1, vC1) ||
                        check_TriangleBoxOverlap(BinCenter, BinHalfSizes, vA2, vB2, vC2)) {
                        numSDsTouched++;
                    }
                }
            }
        }
        numBinsTriTouches[triID] = numSDsTouched;

        // No need to do the following if meshUniversalContact is false
        if (meshUniversalContact) {
            // Register sphere--analytical geometry contacts
            deme::objID_t contact_count = 0;
            // Each triangle should also check if it overlaps with an analytical boundary-type geometry
            for (deme::objID_t objB = 0; objB < simParams->nAnalGM; objB++) {
                deme::bodyID_t objBOwner = objOwner[objB];
                // Grab family number from memory (not jitified: b/c family number can change frequently in a sim)
                unsigned int objFamilyNum = granData->familyID[objBOwner];
                deme::bodyID_t triOwnerID = granData->ownerMesh[triID];
                unsigned int triFamilyNum = granData->familyID[triOwnerID];
                unsigned int maskMatID = locateMaskPair<unsigned int>(triFamilyNum, objFamilyNum);
                // If marked no contact, skip ths iteration
                if (granData->familyMasks[maskMatID] != deme::DONT_PREVENT_CONTACT) {
                    continue;
                }
                double3 ownerXYZ;
                voxelIDToPosition<double, deme::voxelID_t, deme::subVoxelPos_t>(
                    ownerXYZ.x, ownerXYZ.y, ownerXYZ.z, granData->voxelID[objBOwner], granData->locX[objBOwner],
                    granData->locY[objBOwner], granData->locZ[objBOwner], _nvXp2_, _nvYp2_, _voxelSize_, _l_);
                const float ownerOriQw = granData->oriQw[objBOwner];
                const float ownerOriQx = granData->oriQx[objBOwner];
                const float ownerOriQy = granData->oriQy[objBOwner];
                const float ownerOriQz = granData->oriQz[objBOwner];
                float objBRelPosX = objRelPosX[objB];
                float objBRelPosY = objRelPosY[objB];
                float objBRelPosZ = objRelPosZ[objB];
                float objBRotX = objRotX[objB];
                float objBRotY = objRotY[objB];
                float objBRotZ = objRotZ[objB];
                applyOriQToVector3<float, deme::oriQ_t>(objBRelPosX, objBRelPosY, objBRelPosZ, ownerOriQw, ownerOriQx,
                                                        ownerOriQy, ownerOriQz);
                applyOriQToVector3<float, deme::oriQ_t>(objBRotX, objBRotY, objBRotZ, ownerOriQw, ownerOriQx,
                                                        ownerOriQy, ownerOriQz);
                double3 objBPosXYZ = ownerXYZ + make_double3(objBRelPosX, objBRelPosY, objBRelPosZ);

                double3 nodeA, nodeB, nodeC;
                nodeA = to_real3<float3, double3>(vA1);
                nodeB = to_real3<float3, double3>(vB1);
                nodeC = to_real3<float3, double3>(vC1);
                deme::contact_t contact_type = checkTriEntityOverlap<double3>(
                    nodeA, nodeB, nodeC, objType[objB], objBPosXYZ, make_float3(objBRotX, objBRotY, objBRotZ),
                    objSize1[objB], objSize2[objB], objSize3[objB], objNormal[objB], granData->marginSize[objBOwner]);
                if (contact_type == deme::NOT_A_CONTACT) {
                    nodeA = to_real3<float3, double3>(vA2);
                    nodeB = to_real3<float3, double3>(vB2);
                    nodeC = to_real3<float3, double3>(vC2);
                    contact_type = checkTriEntityOverlap<double3>(nodeA, nodeB, nodeC, objType[objB], objBPosXYZ,
                                                                  make_float3(objBRotX, objBRotY, objBRotZ),
                                                                  objSize1[objB], objSize2[objB], objSize3[objB],
                                                                  objNormal[objB], granData->marginSize[objBOwner]);
                }
                // Unlike the sphere-X contact case, we do not test against family extra margin here. This may result in
                // more fake contact pairs, but the efficiency in the mesh-based particle case is not our top priority
                // yet.
                if (contact_type == deme::TRIANGLE_ANALYTICAL_CONTACT) {
                    contact_count++;
                }
            }
            numAnalGeoTriTouches[triID] = contact_count;
        }
    }
}

__global__ void populateBinTriangleTouchingPairs(deme::DEMSimParams* simParams,
                                                 deme::DEMDataKT* granData,
                                                 deme::binsTriangleTouchPairs_t* numBinsTriTouchesScan,
                                                 deme::binsTriangleTouchPairs_t* numAnalGeoTriTouchesScan,
                                                 deme::binID_t* binIDsEachTriTouches,
                                                 deme::bodyID_t* triIDsEachBinTouches,
                                                 float3* nodeA1,
                                                 float3* nodeB1,
                                                 float3* nodeC1,
                                                 float3* nodeA2,
                                                 float3* nodeB2,
                                                 float3* nodeC2,
                                                 deme::bodyID_t* idGeoA,
                                                 deme::bodyID_t* idGeoB,
                                                 deme::contact_t* contactType,
                                                 bool meshUniversalContact) {
    deme::bodyID_t triID = blockIdx.x * blockDim.x + threadIdx.x;
    if (triID < simParams->nTriGM) {
        // 3 vertices of the triangle
        float3 vA1, vB1, vC1, vA2, vB2, vC2;
        deme::binID_t L1[3], L2[3], U1[3], U2[3];
        figureOutNodeAndBoundingBox(simParams, granData, triID, vA1, vB1, vC1, L1, U1, nodeA1[triID], nodeB1[triID],
                                    nodeC1[triID]);
        figureOutNodeAndBoundingBox(simParams, granData, triID, vA2, vB2, vC2, L2, U2, nodeA2[triID], nodeB2[triID],
                                    nodeC2[triID]);
        L1[0] = DEME_MIN(L1[0], L2[0]);
        L1[1] = DEME_MIN(L1[1], L2[1]);
        L1[2] = DEME_MIN(L1[2], L2[2]);
        U1[0] = DEME_MAX(U1[0], U2[0]);
        U1[1] = DEME_MAX(U1[1], U2[1]);
        U1[2] = DEME_MAX(U1[2], U2[2]);

        deme::binsTriangleTouchPairs_t myReportOffset = numBinsTriTouchesScan[triID];
        // In case this sweep does not agree with the previous one, we need to intercept such potential segfaults
        const deme::binsTriangleTouchPairs_t myReportOffset_end = numBinsTriTouchesScan[triID + 1];

        // Triangle may span a collection of bins...
        float BinCenter[3];
        float BinHalfSizes[3];
        BinHalfSizes[0] = simParams->binSize / 2. + DEME_BIN_ENLARGE_RATIO_FOR_FACETS * simParams->binSize;
        BinHalfSizes[1] = simParams->binSize / 2. + DEME_BIN_ENLARGE_RATIO_FOR_FACETS * simParams->binSize;
        BinHalfSizes[2] = simParams->binSize / 2. + DEME_BIN_ENLARGE_RATIO_FOR_FACETS * simParams->binSize;
        for (deme::binID_t i = L1[0]; i <= U1[0]; i++) {
            for (deme::binID_t j = L1[1]; j <= U1[1]; j++) {
                for (deme::binID_t k = L1[2]; k <= U1[2]; k++) {
                    if (myReportOffset >= myReportOffset_end) {
                        continue;  // Don't step on the next triangle's domain
                    }
                    BinCenter[0] = simParams->binSize * i + simParams->binSize / 2.;
                    BinCenter[1] = simParams->binSize * j + simParams->binSize / 2.;
                    BinCenter[2] = simParams->binSize * k + simParams->binSize / 2.;

                    if (check_TriangleBoxOverlap(BinCenter, BinHalfSizes, vA1, vB1, vC1) ||
                        check_TriangleBoxOverlap(BinCenter, BinHalfSizes, vA2, vB2, vC2)) {
                        binIDsEachTriTouches[myReportOffset] =
                            binIDFrom3Indices<deme::binID_t>(i, j, k, simParams->nbX, simParams->nbY, simParams->nbZ);
                        triIDsEachBinTouches[myReportOffset] = triID;
                        myReportOffset++;
                    }
                }
            }
        }
        // This can happen for like 1 in 10^9 chance, for the tri--bin contact algorithm has stochasticity on GPU
        for (; myReportOffset < myReportOffset_end; myReportOffset++) {
            binIDsEachTriTouches[myReportOffset] = deme::NULL_BINID;
            triIDsEachBinTouches[myReportOffset] = triID;
        }

        // No need to do the following if meshUniversalContact is false
        if (meshUniversalContact) {
            deme::binsTriangleTouchPairs_t myTriGeoReportOffset = numAnalGeoTriTouchesScan[triID];
            deme::binsTriangleTouchPairs_t myTriGeoReportOffset_end = numAnalGeoTriTouchesScan[triID + 1];
            for (deme::objID_t objB = 0; objB < simParams->nAnalGM; objB++) {
                deme::bodyID_t objBOwner = objOwner[objB];
                // Grab family number from memory (not jitified: b/c family number can change frequently in a sim)
                unsigned int objFamilyNum = granData->familyID[objBOwner];
                deme::bodyID_t triOwnerID = granData->ownerMesh[triID];
                unsigned int triFamilyNum = granData->familyID[triOwnerID];
                unsigned int maskMatID = locateMaskPair<unsigned int>(triFamilyNum, objFamilyNum);
                // If marked no contact, skip ths iteration
                if (granData->familyMasks[maskMatID] != deme::DONT_PREVENT_CONTACT) {
                    continue;
                }
                double3 ownerXYZ;
                voxelIDToPosition<double, deme::voxelID_t, deme::subVoxelPos_t>(
                    ownerXYZ.x, ownerXYZ.y, ownerXYZ.z, granData->voxelID[objBOwner], granData->locX[objBOwner],
                    granData->locY[objBOwner], granData->locZ[objBOwner], _nvXp2_, _nvYp2_, _voxelSize_, _l_);
                const float ownerOriQw = granData->oriQw[objBOwner];
                const float ownerOriQx = granData->oriQx[objBOwner];
                const float ownerOriQy = granData->oriQy[objBOwner];
                const float ownerOriQz = granData->oriQz[objBOwner];
                float objBRelPosX = objRelPosX[objB];
                float objBRelPosY = objRelPosY[objB];
                float objBRelPosZ = objRelPosZ[objB];
                float objBRotX = objRotX[objB];
                float objBRotY = objRotY[objB];
                float objBRotZ = objRotZ[objB];
                applyOriQToVector3<float, deme::oriQ_t>(objBRelPosX, objBRelPosY, objBRelPosZ, ownerOriQw, ownerOriQx,
                                                        ownerOriQy, ownerOriQz);
                applyOriQToVector3<float, deme::oriQ_t>(objBRotX, objBRotY, objBRotZ, ownerOriQw, ownerOriQx,
                                                        ownerOriQy, ownerOriQz);
                double3 objBPosXYZ = ownerXYZ + make_double3(objBRelPosX, objBRelPosY, objBRelPosZ);

                double3 nodeA, nodeB, nodeC;
                nodeA = to_real3<float3, double3>(vA1);
                nodeB = to_real3<float3, double3>(vB1);
                nodeC = to_real3<float3, double3>(vC1);
                deme::contact_t contact_type = checkTriEntityOverlap<double3>(
                    nodeA, nodeB, nodeC, objType[objB], objBPosXYZ, make_float3(objBRotX, objBRotY, objBRotZ),
                    objSize1[objB], objSize2[objB], objSize3[objB], objNormal[objB], granData->marginSize[objBOwner]);
                if (contact_type == deme::NOT_A_CONTACT) {
                    nodeA = to_real3<float3, double3>(vA2);
                    nodeB = to_real3<float3, double3>(vB2);
                    nodeC = to_real3<float3, double3>(vC2);
                    contact_type = checkTriEntityOverlap<double3>(nodeA, nodeB, nodeC, objType[objB], objBPosXYZ,
                                                                  make_float3(objBRotX, objBRotY, objBRotZ),
                                                                  objSize1[objB], objSize2[objB], objSize3[objB],
                                                                  objNormal[objB], granData->marginSize[objBOwner]);
                }
                // Unlike the sphere-X contact case, we do not test against family extra margin here.
                if (contact_type == deme::TRIANGLE_ANALYTICAL_CONTACT) {
                    idGeoA[myTriGeoReportOffset] = triID;
                    idGeoB[myTriGeoReportOffset] = (deme::bodyID_t)objB;
                    contactType[myTriGeoReportOffset] = contact_type;
                    myTriGeoReportOffset++;
                    if (myTriGeoReportOffset >= myTriGeoReportOffset_end) {
                        return;  // Don't step on the next triangle's domain
                    }
                }
            }
            // Take care of potentially unfilled slots in the report
            for (; myTriGeoReportOffset < myTriGeoReportOffset_end; myTriGeoReportOffset++) {
                contactType[myTriGeoReportOffset] = deme::NOT_A_CONTACT;
            }
        }
    }
}

__global__ void mapTriActiveBinsToSphActiveBins(deme::binID_t* activeBinIDsForTri,
                                                deme::binID_t* activeBinIDs,
                                                deme::binID_t* mapTriActBinToSphActBin,
                                                size_t numActiveBinsForTri,
                                                size_t numActiveBinsForSph) {
    size_t threadID = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadID < numActiveBinsForTri) {
        deme::binID_t binID = activeBinIDsForTri[threadID];
        deme::binID_t indexInOther;
        bool found = cuda_binary_search<deme::binID_t, deme::binID_t>(activeBinIDs, binID, 0, numActiveBinsForSph - 1,
                                                                      indexInOther);
        if (found) {
            mapTriActBinToSphActBin[threadID] = indexInOther;
        } else {
            mapTriActBinToSphActBin[threadID] = deme::NULL_BINID;
        }
    }
}
