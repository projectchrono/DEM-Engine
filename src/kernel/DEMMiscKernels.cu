#include "hip/hip_runtime.h"
// DEM misc. kernels
#include <DEM/Defines.h>
#include <DEMHelperKernels.cuh>

__global__ void markOwnerToChange(deme::notStupidBool_t* idBool,
                                  float* ownerFactors,
                                  deme::bodyID_t* dIDs,
                                  float* dFactors,
                                  size_t n) {
    size_t myID = blockIdx.x * blockDim.x + threadIdx.x;
    if (myID < n) {
        deme::bodyID_t myOwner = dIDs[myID];
        float myFactor = dFactors[myID];
        idBool[myOwner] = 1;
        ownerFactors[myOwner] = myFactor;
    }
}

template <typename DEMData>
__global__ void modifyComponents(DEMData* granData, deme::notStupidBool_t* idBool, float* factors, size_t n) {
    size_t sphereID = blockIdx.x * blockDim.x + threadIdx.x;
    if (sphereID < n) {
        // Get my owner ID
        deme::bodyID_t myOwner = granData->ownerClumpBody[sphereID];
        // If not marked, we have nothing to do
        if (idBool[myOwner]) {
            float factor = factors[myOwner];
            // Expand radius and relPos
            granData->relPosSphereX[sphereID] *= factor;
            granData->relPosSphereY[sphereID] *= factor;
            granData->relPosSphereZ[sphereID] *= factor;
            granData->radiiSphere[sphereID] *= factor;
        }
    }
}

__global__ void computeMarginFromAbsv(deme::DEMSimParams* simParams,
                                      deme::DEMDataKT* granData,
                                      float* ts,
                                      unsigned int* maxDrift,
                                      size_t n) {
    size_t ownerID = blockIdx.x * blockDim.x + threadIdx.x;
    if (ownerID < n) {
        float absv = granData->marginSize[ownerID];
        unsigned int my_family = granData->familyID[ownerID];
        if (!isfinite(absv)) {
            // May produce messy error messages, but it's still good to know what entities went wrong
            DEME_ABORT_KERNEL(
                "Absolute velocity for ownerID %llu is infinite (and it's a worse version of "
                "max-velocity-exceeded-allowance).\n",
                static_cast<unsigned long long>(ownerID));
        }
        if (absv > simParams->approxMaxVel) {
            absv = simParams->approxMaxVel;
        }
        // User-specified extra margin also needs to be added here. This marginSize is used for bin--sph or bin--tri
        // contacts but not entirely the same as the one used for sph--sph or sph--tri contacts, since the latter is
        // stricter.
        granData->marginSize[ownerID] =
            (double)(absv * simParams->expSafetyMulti + simParams->expSafetyAdder) * (*ts) * (*maxDrift) +
            granData->familyExtraMarginSize[my_family];
    }
}

__global__ void fillMarginValues(deme::DEMSimParams* simParams, deme::DEMDataKT* granData, size_t n) {
    size_t ownerID = blockIdx.x * blockDim.x + threadIdx.x;
    if (ownerID < n) {
        unsigned int my_family = granData->familyID[ownerID];
        granData->marginSize[ownerID] = simParams->beta + granData->familyExtraMarginSize[my_family];
    }
}
