#include "hip/hip_runtime.h"
// DEM force calculation strategies, modifiable

if (overlapDepth > 0) {
    // Material properties
    float E_cnt, CoR_cnt;
    {
        matProxy2ContactParam<float>(E_cnt, E[bodyAMatType], nu[bodyAMatType], E[bodyBMatType], nu[bodyBMatType]);
        // CoR is pair-wise, so obtain it this way
        CoR_cnt = CoR[bodyAMatType][bodyBMatType];
    }

    float3 rotVelCPA, rotVelCPB;
    {
        // We also need the relative velocity between A and B in global frame to use in the damping terms
        // To get that, we need contact points' rotational velocity in GLOBAL frame
        // This is local rotational velocity (the portion of linear vel contributed by rotation)
        rotVelCPA = cross(ARotVel, locCPA);
        rotVelCPB = cross(BRotVel, locCPB);
        // This is mapping from local rotational velocity to global
        applyOriQToVector3<float, deme::oriQ_t>(rotVelCPA.x, rotVelCPA.y, rotVelCPA.z, AOriQ.w, AOriQ.x, AOriQ.y,
                                                AOriQ.z);
        applyOriQToVector3<float, deme::oriQ_t>(rotVelCPB.x, rotVelCPB.y, rotVelCPB.z, BOriQ.w, BOriQ.x, BOriQ.y,
                                                BOriQ.z);
    }

    // The (total) relative linear velocity of A relative to B
    const float3 velB2A = (ALinVel + rotVelCPA) - (BLinVel + rotVelCPB);
    const float projection = dot(velB2A, B2A);

    const float mass_eff = (AOwnerMass * BOwnerMass) / (AOwnerMass + BOwnerMass);

    // Contact radius (radial distance from contact center axis) called cnt_rad, computed from area
    const float cnt_rad = sqrtf(overlapArea / deme::PI);
    const float Sn = 2.f * E_cnt * cnt_rad;

    const float loge = (CoR_cnt < DEME_TINY_FLOAT) ? log(DEME_TINY_FLOAT) : log(CoR_cnt);
    const float beta = loge / sqrt(loge * loge + deme::PI_SQUARED);

    const float k_n = (2.f / 3.f) * Sn;
    const float gamma_n = (2.f * sqrtf(5.f / 6.f)) * beta * sqrtf(Sn * mass_eff);

    force += (k_n * overlapDepth + gamma_n * projection) * B2A;
    printf("Area is %f, cnt_rad is %f, Sn is %f, k_n is %f, gamma_n is %f, force is %f %f %f\n", overlapArea, cnt_rad,
           Sn, k_n, gamma_n, force.x, force.y, force.z);
}
